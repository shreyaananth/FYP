#include "hip/hip_runtime.h"
#include <stdio.h>
#include "runtime.cuh"
#include <stdint.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <time.h>
#include <sys/time.h>
#include <fcntl.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <unistd.h>
#include <stdlib.h>
using namespace std;
int steps = 2048 * 80;
int threadnum = 128;
int evict_tlb_size = 5120;

__device__ void sleep_overflow(int64_t num_cycles)
{
	int64_t cycles = 0;
	int64_t start = clock64();
	while (cycles < num_cycles)
	{
		cycles = clock64() - start;
	}
}

double GetMSTime(void)
{
	struct timeval stNowTime;

	gettimeofday(&stNowTime, NULL);

	return (1.0 * (stNowTime.tv_sec) * 1000 + 1.0 * (stNowTime.tv_usec) / 1000);
}
__device__ __inline__ int ld_gbl_cg(unsigned int *addr)
{
	unsigned int return_value;
	asm("ld.global.cg.s32 %0, [%1];"
		: "=r"(return_value)
		: "l"(addr));
	return return_value;
}

void add_diff(unsigned int *a, unsigned int *b)
{
	unsigned int *tmp = NULL;
	if (a > b)
	{
		tmp = a;
		a = b;
		b = tmp;
	}
	printf("add1=%p,add2=%p,address diff =%0.2f MB\n", a, b, (double)((unsigned long)b - (unsigned long)a) / 1024 / 1024);
}
__global__ void tlb2_attack2(unsigned int **my_array, int steps, int *compiler_array, int memNum, Policy con)
{

  __shared__ int bidx[1];
  __shared__ int bidy[1];
  __shared__ int miss_num[1];
  if (!runable_retreat(bidx, bidy, miss_num, con))
    return;

	int k;
	unsigned int j = 0;
	unsigned int w = threadIdx.x % memNum;

	unsigned int compile = 0;
#pragma unroll 1
	for (k = 0; k < steps; k++)
	{
		j = ld_gbl_cg(my_array[w] + j);
		asm volatile("membar.cta;");
		w = (w + 1) % memNum;
		compile += j;
	}

	compiler_array[blockDim.x * blockIdx.x + threadIdx.x] = j;
	compiler_array[blockDim.x * blockIdx.x + threadIdx.x + 1] = my_array[w][j];
	compiler_array[blockDim.x * blockIdx.x + threadIdx.x + 2] = compile;
}

void tlb2_attack(int attack_tlbsize, int attack_stride, const int &app_id)
{
	unsigned int devNo = 0;
	hipSetDevice(devNo);
	int memNum = attack_tlbsize / attack_stride;
	printf("memNum=%d\n", memNum);
	unsigned int *t_arr[memNum];
	unsigned int *tmp[15 * memNum];
	unsigned int **d_arr;
	hipMalloc((void **)&d_arr, sizeof(unsigned int *) * memNum);
	for (int i = 0; i < memNum; i++)
	{
		hipMalloc((void **)&(t_arr[i]), sizeof(unsigned int) * 2 * 1024 * 256);
		for (int k = 0; k < 15; k++)
		{
			hipMalloc((void **)&tmp[i * 15 + k], sizeof(unsigned int) * 2 * 1024 * 256);
		}
		hipMemset(t_arr[i], 0, sizeof(unsigned int) * 2 * 1024 * 256);
	}
	for (int k = 0; k < 15 * memNum; k++)
	{ //release last round memory
		hipFree(tmp[k]);
	}
	hipMemcpy(d_arr, t_arr, sizeof(unsigned int *) * memNum, hipMemcpyHostToDevice);
	int SMcount = 0;
	hipDeviceProp_t props;
	hipGetDeviceProperties(&props, devNo);
	cout << "#" << props.name << ": cuda " << props.major << "." << props.minor << endl;
	SMcount = props.multiProcessorCount;
	printf("ATTACK PROGRAM\n");
	printf("SMcount=%d\n", SMcount);
	int *compiler_array;
	hipMalloc((void **)&compiler_array, sizeof(int) * (SMcount * 3 * threadnum));
	hipMemset(compiler_array, 0, sizeof(int) * SMcount * threadnum * 3);
	double t1, t2;
	fprintf(stderr, "attack_tlbsize=%d,attack_stride=%d,steps=%d,threadnum=%d\n", attack_tlbsize, attack_stride, steps, threadnum);


App app(app_id);
  
dim3 grid(SMcount, 1, 1);
    dim3 block(threadnum, 1, 1);

  Policy task1 = dispatcher(app, grid, block);

	t1 = GetMSTime();

	tlb2_attack2<<<grid, block>>>(d_arr, steps, compiler_array, memNum, task1);

	hipDeviceSynchronize();
	t2 = GetMSTime();
  task_destroy(task1);

	fprintf(stderr, "GPU-concatenation-time: %4.2lf ms\n", (t2 - t1));
	for (int i = 0; i < memNum; i++)
	{
		hipFree(t_arr[i]);
	}
	hipFree(d_arr);
}

int main(int argc, char **argv)
{
	if(argc==5){

		threadnum = atoi(argv[2]);
		steps = atoi(argv[3]);
		evict_tlb_size = atoi(argv[4]);

	}
	int app_id = atoi(argv[1]);
	tlb2_attack(evict_tlb_size, 32, app_id);
	return 0;
}
