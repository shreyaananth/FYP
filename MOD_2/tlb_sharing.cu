#include "hip/hip_runtime.h"
/*-
 * Copyright (c) 2017 Tomas Karnagel and Matthias Werner
 * All rights reserved.
 *
 * Redistribution and use in source and binary forms, with or without
 * modification, are permitted provided that the following conditions
 * are met:
 * 1. Redistributions of source code must retain the above copyright
 *    notice, this list of conditions and the following disclaimer.
 * 2. Redistributions in binary form must reproduce the above copyright
 *    notice, this list of conditions and the following disclaimer in the
 *    documentation and/or other materials provided with the distribution.
 *
 * THIS SOFTWARE IS PROVIDED BY THE AUTHOR AND CONTRIBUTORS ``AS IS'' AND
 * ANY EXPRESS OR IMPLIED WARRANTIES, INCLUDING, BUT NOT LIMITED TO, THE
 * IMPLIED WARRANTIES OF MERCHANTABILITY AND FITNESS FOR A PARTICULAR PURPOSE
 * ARE DISCLAIMED.  IN NO EVENT SHALL THE AUTHOR OR CONTRIBUTORS BE LIABLE
 * FOR ANY DIRECT, INDIRECT, INCIDENTAL, SPECIAL, EXEMPLARY, OR CONSEQUENTIAL
 * DAMAGES (INCLUDING, BUT NOT LIMITED TO, PROCUREMENT OF SUBSTITUTE GOODS
 * OR SERVICES; LOSS OF USE, DATA, OR PROFITS; OR BUSINESS INTERRUPTION)
 * HOWEVER CAUSED AND ON ANY THEORY OF LIABILITY, WHETHER IN CONTRACT, STRICT
 * LIABILITY, OR TORT (INCLUDING NEGLIGENCE OR OTHERWISE) ARISING IN ANY WAY
 * OUT OF THE USE OF THIS SOFTWARE, EVEN IF ADVISED OF THE POSSIBILITY OF
 * SUCH DAMAGE.
 *
 */

#include "helper.h"
#include <stdio.h>
#include <assert.h>
#include <iostream>
#include <iomanip>
#include <fstream>
#include <vector>
#include <limits>
#include <algorithm>
#include <sys/time.h>
#include <sys/stat.h>
#include <sys/types.h>
#include <string>

#include <hip/hip_runtime.h>


using namespace std;

enum Metric { METRIC_AVG, METRIC_MIN };

// --------------------------------- GPU Kernel ------------------------------

template<bool DISRUPT, bool GET_DURATION>
static __global__
void tlb_latency_with_disruptor(unsigned int * hashtable,
                                unsigned hashtable_count,
                                unsigned iterations,
                                unsigned stride_count,
                                unsigned offset,
                                int smid0,
                                int smxxx)
{
    extern __shared__ unsigned duration[]; // shared memory should be large enough to fill one SM

    unsigned smid;
    asm("mov.u32 %0, %smid;" : "=r"(smid) );

    if(!(DISRUPT || smid==smid0)) // only take 1st SM in non-disrupting mode
        return;
    if(DISRUPT && smid!=smxxx) // only SMxxx does run in disrupting mode
        return;
    if(threadIdx.x!=0)
        return;

    unsigned long start;
    unsigned int sum = 0;
    unsigned int pos = DISRUPT ? (stride_count*iterations + offset) % hashtable_count : offset;
    sum += pos; // ensure pos is set before entering loop
    for (unsigned int i = 0; i < iterations; i++) {
        start = clock64();
        pos = hashtable[pos];
        sum += pos; // ensure pos is set before taking clock
        duration[i] = static_cast<unsigned>(clock64()-start);
    }
    if(sum == 0)
        hashtable[hashtable_count+1] = sum;
    if(GET_DURATION && smid==smid0) { // only store durations one time
        for (unsigned int i = 0; i < iterations; i++) {
            hashtable[hashtable_count+2+i] = duration[i];
        }
    }
}


// --------------------------------- main part ------------------------------
int main(int argc, char **argv)
{
    unsigned int devNo = 0;
    Metric metric = METRIC_AVG;
    // ------------- handle inputs ------------
      
    if (argc < 3) {
        cerr << "usage: " << argv[0] << " stride_KB iterations device_No=0" << endl;
        return 0;
    }

    int stride_KB = atoi(argv[1]);
    int iterations = atoi(argv[2]);

    if (argc > 3)
        devNo = atoi(argv[3]);

    if (argc > 4 && strcmp(argv[4],"1")==0)
        metric = METRIC_MIN;

    // --------------- init CUDA ---------
    int devCount;
    int SMcount =  0;
    CHECK_CUDA(hipGetDeviceCount(&devCount));
    
    // check Dev Count
    if (devNo >= devCount){
        cerr << "Can not choose Dev " << devNo << ", only " << devCount << " GPUs " << endl;
        exit(0);
    }
    CHECK_CUDA(hipSetDevice(devNo));
        
    hipDeviceProp_t props;
    CHECK_CUDA(hipGetDeviceProperties(&props, devNo));
    cout << "#" << props.name << ": cuda " << props.major << "." << props.minor << endl;
    cout << ", reduction mode: " << (metric == METRIC_MIN ? "min" : "avg") << endl;
    SMcount = props.multiProcessorCount;


    // --------------- setup input data ---------
    size_t hashtable_size_MB = ((iterations+1) * stride_KB * 2) / 1024;
    
    
    CHECK_CUDA(hipDeviceReset());
      
    unsigned int * hashtable;
    unsigned* hduration = new unsigned [iterations];
    size_t N = hashtable_size_MB*1048576llu/sizeof(unsigned int);
     
    CHECK_CUDA(hipMalloc(&hashtable, hashtable_size_MB*1048576llu+(iterations+2llu)*sizeof(unsigned int)));
      
    // init data
    unsigned int* hdata = new unsigned int[N+1];
    size_t stride_count = stride_KB*1024llu/sizeof(unsigned);
    for(size_t t=0; t<N; ++t) {
        hdata[t] = ( t+stride_count ) % N;
    }
    hdata[N] = 0;
    CHECK_CUDA(hipMemcpy(hashtable, hdata, (N+1)*sizeof(unsigned), hipMemcpyHostToDevice));
    delete[] hdata;

    // alloc output space
    double ** results = new double * [SMcount];
    for(int i = 0; i < SMcount; ++i){
        results[i] = new double[SMcount];
        memset(results[i], 0, sizeof(double) * SMcount);
    }
    
    
    // --------------- test all SMx to SMy combinations ---------

    for (int smid0 = 0; smid0 < SMcount; smid0++){
        for  (int smxxx = 0; smxxx < SMcount; smxxx++) {

            CHECK_CUDA(hipDeviceSynchronize());
                
            // fill TLB
            tlb_latency_with_disruptor<false, false><<<2*SMcount, 1, iterations*sizeof(unsigned)>>>(hashtable, N,  iterations, stride_count, 0, smid0,smxxx);
              
            // disrupt TLB if TLB is shared
            tlb_latency_with_disruptor<true, false><<<2*SMcount, 1, iterations*sizeof(unsigned)>>>(hashtable, N,  iterations, stride_count, 0, smid0,smxxx);
              
            // check if values in TLB
            tlb_latency_with_disruptor<false, true><<<2*SMcount, 1, iterations*sizeof(unsigned)>>>(hashtable, N,  iterations, stride_count, 0, smid0,smxxx);
             
             
            CHECK_LAST( "Kernel failed." );
            CHECK_CUDA(hipDeviceSynchronize());
    
            
            // get needed cycles
            CHECK_CUDA(hipMemcpy(hduration, hashtable+N+2, iterations*sizeof(unsigned), hipMemcpyDeviceToHost));

            if(metric==METRIC_AVG) {
                double avgc=0;
                for(int b=0; b<iterations;++b) {
                    avgc+=hduration[b];
                }
                results[smid0][smxxx] =  avgc/iterations;
            } else {
                unsigned int minc=std::numeric_limits<unsigned int>::max();
                for(int b=0; b<iterations;++b) {
                    minc = std::min(hduration[b],minc);
                }
                results[smid0][smxxx] =  minc;
            }
        }
    }
    
    CHECK_CUDA(hipFree(hashtable));
    delete[] hduration;
    
 
    // ---------------output handling ---------
 
    cout << "#----------- absolute values ---------------" << endl;
 
    cout << "#   ";
    for (unsigned int steps = 0; steps < SMcount; steps++)
        cout << setfill('0') << setw(3) << steps << " ";
    cout << endl;    
    
    for(unsigned int y = 0; y < SMcount; y++){
        cout << setfill('0') << setw(3) << y << " ";
        for(unsigned int x = 0; x < SMcount; x++){
            cout << (unsigned int) (results[y][x]);
            cout << " ";
        }
        cout << endl;
    }
    
    cout << "#----------- which SMs share TLB ---------------" << endl;
    
    cout << "#  ";
    for (unsigned int steps = 0; steps < SMcount; steps++)
        cout << setfill('0') << setw(2) << steps << " ";
    cout << endl;    
    
    for(unsigned int y = 0; y < SMcount; y++){
        cout << setfill('0') << setw(2) << y << " ";

        double avg = 0;
        for(unsigned int x = 0; x < SMcount; x++) {
            avg += (results[y][x]);
        }
        // build average and add some buffer
        avg = (avg / SMcount)+3;

        for(unsigned int x = 0; x < SMcount; x++){
            if (results[y][x] > avg) {
                double avgt = 0;
                for(unsigned int xt = 0; xt < SMcount; xt++) {
                    avgt += (results[x][xt]);
                }
                // build average and add some buffer
                avgt = (avgt / SMcount)+3;
                if (results[x][y] > avgt)
                    cout << ".X ";
                else
                    cout << ".. ";
            } else
                cout << ".. ";
        }
        cout << " [" << avg << "]";
        cout << endl;
    }

 
    CHECK_CUDA(hipDeviceReset());
    return 0;
}
